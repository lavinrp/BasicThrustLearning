#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <thrust\device_vector.h>
#include <thrust\host_vector.h>
#include <thrust\device_ptr.h>
#include <thrust\memory.h>
#include <thrust\copy.h>

#include <stdio.h>

#include <iostream>



struct isEven {

	__device__ __host__
	bool operator()(int x) 
	{
		return (x % 2) == 0;
	}

};

__device__
int getGlobalIdx_3D_3D() 
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x
		+ gridDim.x * gridDim.y * blockIdx.z;
	int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
		+ (threadIdx.z * (blockDim.x * blockDim.y))
		+ (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__global__
void addOne( int const * const __restrict__ inputData, int* __restrict__ outputData, size_t dataSize) 
{
	int thid = getGlobalIdx_3D_3D();
	if (thid >= dataSize) 
	{
		return;
	}

	outputData[thid] = inputData[thid] + 1;
}

void addOneExample(const int DATA_SIZE) 
{
	thrust::host_vector<int> h_inputData(DATA_SIZE);
	std::cout << "input data:" << std::endl;
	for (size_t i = 0; i < DATA_SIZE; i++) {
		h_inputData[i] = i;
		std::cout << i << std::endl;
	}
	std::cout << std::endl << std::endl;

	//move data to device
	thrust::device_vector<int> d_inputData = h_inputData;

	//allocate output data array
	thrust::device_vector<int> d_outputData(DATA_SIZE);

	addOne <<<2, 5>>>(thrust::raw_pointer_cast(d_inputData.data()), thrust::raw_pointer_cast(d_outputData.data()), DATA_SIZE);

	//move data back to host
	thrust::host_vector<int> h_outputData = d_outputData;

	std::cout << "output data: " << std::endl;
	for (size_t i = 0; i < h_outputData.size(); i++) {
		std::cout << h_outputData[i] << std::endl;
	}

	std::cout << std::endl << std::endl;
}

void filterExample(const int DATA_SIZE) 
{
	thrust::host_vector<int> h_inputData(DATA_SIZE);
	std::cout << "input data:" << std::endl;
	for (size_t i = 0; i < DATA_SIZE; i++) {
		h_inputData[i] = i;
		std::cout << i << std::endl;
	}
	std::cout << std::endl << std::endl;

	//move data to device
	thrust::device_vector<int> d_inputData = h_inputData;

	//allocate output data array
	thrust::device_vector<int> d_outputData(DATA_SIZE);

	auto lastCopiedValue = thrust::copy_if(d_inputData.begin(), d_inputData.end(), d_outputData.begin(), isEven());

	//move data back to host
	thrust::host_vector<int> h_outputData = d_outputData;

	std::cout << "output data: " << std::endl;
	for (int i = 0; i < std::distance(d_outputData.begin(), lastCopiedValue); i++) 
	{
		std::cout << h_outputData[i] << std::endl;
	}

	std::cout << std::endl << std::endl;

}

int main()
{

	//add one to data

	//create input data
	const int DATA_SIZE = 5;
	
	std::cout << "starting addOne" << std::endl;
	addOneExample(DATA_SIZE);

	std::cout << "press ENTER to continue...";
	std::cin.get();

	std::cout << "starting filter" << std::endl;
	filterExample(DATA_SIZE);

	std::cout << "press ENTER to exit...";
	std::cin.get();

	//filter data based on even odd


}

